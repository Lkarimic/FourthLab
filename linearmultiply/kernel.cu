/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void mysgemm(int m, int n, int k, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A x B
     *   where A is a (m x k) matrix
     *   where B is a (k x n) matrix
     *   where C is a (m x n) matrix
     *
     ********************************************************************/
int Row = blockIdx.y*blockDim.y+threadIdx.y;
int Col = blockIdx.x*blockDim.x+threadIdx.x;
 if ((Row < m) && (Col < n)) {
   float Pvalue = 0;
   for (int i = 0; i< k; ++i)
   Pvalue += A[Row*k+i] * B[i*n+Col];
    C[Row*n+Col] = Pvalue;
  }

}

void basicSgemm(char transa, char transb, int m, int n, int k, float alpha, const float *A, int lda, const float *B, int ldb, float beta, float *C, int ldc)
{
    if ((transa != 'N') && (transa != 'n')) {
	printf("unsupported value of 'transa'\n");
    	return;
    }

    if ((transb != 'N') && (transb != 'n')) {
	printf("unsupported value of 'transb'\n");
	return;
    }

    if ((alpha - 1.0f > 1e-10) || (alpha - 1.0f < -1e-10)) {
	printf("unsupported value of alpha\n");
	return;
    }

    if ((beta - 0.0f > 1e-10) || (beta - 0.0f < -1e-10)) {
	printf("unsupported value of beta\n");
	return;
    }

    // Initialize thread block and kernel grid dimensions ---------------------
    const unsigned int BLOCK_SIZE = 16; // Use 16x16 thread block
    dim3 dimGrid(ceil(n/(float)BLOCK_SIZE),(ceil(m/(float)BLOCK_SIZE)), 1 );
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);


    //INSERT CODE HERE to define thread blocks and layout

    // Invoke CUDA kernel -----------------------------------------------------
mysgemm<<<dimGrid, dimBlock>>>(m, n, k, A, B, C);
    //INSERT CODE HERE



}
